
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void ph_dRPA_AmB_sing_kernel(int nO, int nV, int nBas, int nS, 
                                        double *eps, double *ERI, double *AmB) {


    int i, j, a, b;
    int aa, bb;

    long long i_A0, i_A1, i_A2, i_A3;
    long long i_I0, i_I1, i_I2, i_I3;
    long long i_J1, i_J2, i_J3;

    long long nVS;
    long long nBas2, nBas3;

    bool a_eq_b;

    nVS = (long long) nV * (long long) nS;

    nBas2 = (long long) nBas * (long long) nBas;
    nBas3 = nBas2 * (long long) nBas;

    aa = blockIdx.x * blockDim.x + threadIdx.x;
    bb = blockIdx.y * blockDim.y + threadIdx.y;

    while(aa < nV) {
        a = aa + nO;

        i_A0 = (long long) aa * (long long) nS;
        i_I0 = (long long) a * nBas2;

        while(bb < nV) {
            b = bb + nO;

            a_eq_b = a == b;

            i_A1 = i_A0 + (long long) bb;
            i_I1 = i_I0 + (long long) b * (long long) nBas;
            i_J1 = i_I0 + (long long) b * nBas3;

            i = 0;
            while(i < nO) {

                i_A2 = i_A1 + (long long) i * nVS;
                i_I2 = i_I1 + (long long) i;
                i_J2 = i_J1 + (long long) i;
 
                j = 0;
                while(j < nO) {

                    i_A3 = i_A2 + (long long) j * nV;
                    i_I3 = i_I2 + (long long) j * nBas3;
                    i_J3 = i_J2 + (long long) j * (long long) nBas;

                    AmB[i_A3] = 2.0 * (ERI[i_I3] - ERI[i_J3]);
                    if(a_eq_b && (i==j)) {
                        AmB[i_A3] += eps[a] - eps[i];
                    }

                    j ++;
	        } // j

                i ++;
            } // i

            bb += blockDim.y * gridDim.y;
        } // bb

        aa += blockDim.x * gridDim.x;
    } // aa

}





extern "C" void ph_dRPA_AmB_sing(int nO, int nV, int nBas, int nS, double *eps, double *ERI, double *AmB) {


    int sBlocks = 32;
    int nBlocks = (nV + sBlocks - 1) / sBlocks;

    dim3 dimGrid(nBlocks, nBlocks, 1);
    dim3 dimBlock(sBlocks, sBlocks, 1);

    printf("lunching ph_dRPA_AmB_sing_kernel with %dx%d blocks and %dx%d threads/block\n",
        nBlocks, nBlocks, sBlocks, sBlocks);


    ph_dRPA_AmB_sing_kernel<<<dimGrid, dimBlock>>>(nO, nV, nBas, nS, eps, ERI, AmB);

}




